// matrix_multiplication.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMul(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        C[row * N + col] = 0.0f;
        for (int i = 0; i < N; i++) {
            C[row * N + col] += A[row * N + i] * B[i * N + col];
        }
    }
}

int main() {
    int N = 1024; // size of the matrices
    float *A, *B, *C;
    hipMalloc((void **)&A, N * N * sizeof(float));
    hipMalloc((void **)&B, N * N * sizeof(float));
    hipMalloc((void **)&C, N * N * sizeof(float));

    // initialize matrices A and B
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = (float)(i + j);
            B[i * N + j] = (float)(i - j);
        }
    }

    // launch kernel
    dim3 block(16, 16); // blocks of 16x16 threads
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    matrixMul<<<grid, block>>>(A, B, C, N);

    // copy results back to host
    hipMemcpy(C, C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // print results
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // free memory
    hipFree(A);
    hipFree(B);
    hipFree(C);

    return 0;
}

